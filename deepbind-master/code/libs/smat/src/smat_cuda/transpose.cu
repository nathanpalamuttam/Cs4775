#include "hip/hip_runtime.h"
// Copyright (c) 2015, Andrew Delong and Babak Alipanahi All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
// 
// 1. Redistributions of source code must retain the above copyright notice, this
// list of conditions and the following disclaimer.
// 
// 2. Redistributions in binary form must reproduce the above copyright notice,
// this list of conditions and the following disclaimer in the documentation and/or
// other materials provided with the distribution.
// 
// 3. Neither the name of the copyright holder nor the names of its contributors
// may be used to endorse or promote products derived from this software without
// specific prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
// ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 
// Author's note: 
//     This file was distributed as part of the Nature Biotechnology 
//     supplementary software release for DeepBind. Users of DeepBind
//     are encouraged to instead use the latest source code and binaries 
//     for scoring sequences at
//        http://tools.genes.toronto.edu/deepbind/
// 
#include <smat_cuda/cuda_errors.h>
#include <smat_cuda/cuda_context.h>
#include <smat_cuda/launch_util.h>
#include <smat/vm/util/specialization_table.h>
#include <smat/vm/util/specialization_typelists.h>
#include <smat/vm/instruction_db.h>

SM_NAMESPACE_BEGIN

const unsigned c_trans_tile_size = 16;

template <typename T> 
__global__ void kernel_trans(const T* src, T* dst, isize_t n, isize_t m) {
	DECL_KERNEL_VARS;
	unsigned i,j;
	__shared__ T tile[c_trans_tile_size][c_trans_tile_size+1];

	// Read the tile into shared memory.
	i = c_trans_tile_size*by + ty;
	j = c_trans_tile_size*bx + tx;
	if(i < n && j < m)
		tile[ty][tx] = src[m*i+j];

	__syncthreads();

	// Write the tile to global memory in transposed order
	i = c_trans_tile_size*bx + ty;
	j = c_trans_tile_size*by + tx;
	if(i < m && j < n)
		dst[n*i+j] = tile[tx][ty];
}

template <typename T>
struct execute_transpose_typed { // TODO: autotune this
	static void execute(opcode_t opcode, const argument& src, const argument& dst)
	{
		if (src.size() > 0) {
			dim3 bdim(c_trans_tile_size,c_trans_tile_size);
			dim3 gdim(divup((unsigned)src.shape.x,c_trans_tile_size),
					  divup((unsigned)src.shape.y,c_trans_tile_size));
			kernel_trans<<<gdim,bdim,0,thread_cudactx().stream()>>>(src.get<const T*>(),dst.get<T*>(),src.shape.y,src.shape.x);
		}
	}
};

// Use NVIDIA BLAS extensions to do more highly-tuned transpose for float and double types.
// Use CUBLAS for float or double type.
template <>
struct execute_transpose_typed<float> {
	static void execute(opcode_t opcode, const argument& src, const argument& dst)
	{
		float alpha = 1, beta = 0;
		ccb(Sgeam,thread_cudactx().cublas(),HIPBLAS_OP_T,HIPBLAS_OP_T,(int)src.shape.y,(int)src.shape.x,
			&alpha,src.get<const float*>(),(int)src.shape.x,
			&beta ,src.get<const float*>(),(int)src.shape.x,
			dst.get<float*>(),(int)dst.shape.x)
	}
};

template <>
struct execute_transpose_typed<double> {
	static void execute(opcode_t opcode, const argument& src, const argument& dst)
	{
		double alpha = 1, beta = 0;
		ccb(Dgeam,thread_cudactx().cublas(),HIPBLAS_OP_T,HIPBLAS_OP_T,(int)src.shape.y,(int)src.shape.x,
			&alpha,src.get<const double*>(),(int)src.shape.x,
			&beta ,src.get<const double*>(),(int)src.shape.x,
			dst.get<double*>(),(int)dst.shape.x)
	}
};

void execute_transpose(opcode_t opcode, const argument& src, const argument& dst)
{
	DECL_SPECIALIZATION_TABLE(T_G,execute_fn2,execute_transpose_typed);
	specialization_table(src.dtype)(opcode,src,dst);
}

SM_NAMESPACE_END
