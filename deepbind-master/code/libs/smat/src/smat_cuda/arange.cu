#include "hip/hip_runtime.h"
// Copyright (c) 2015, Andrew Delong and Babak Alipanahi All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
// 
// 1. Redistributions of source code must retain the above copyright notice, this
// list of conditions and the following disclaimer.
// 
// 2. Redistributions in binary form must reproduce the above copyright notice,
// this list of conditions and the following disclaimer in the documentation and/or
// other materials provided with the distribution.
// 
// 3. Neither the name of the copyright holder nor the names of its contributors
// may be used to endorse or promote products derived from this software without
// specific prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
// ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 
// Author's note: 
//     This file was distributed as part of the Nature Biotechnology 
//     supplementary software release for DeepBind. Users of DeepBind
//     are encouraged to instead use the latest source code and binaries 
//     for scoring sequences at
//        http://tools.genes.toronto.edu/deepbind/
// 
#include <smat_cuda/cuda_errors.h>
#include <smat_cuda/cuda_context.h>
#include <smat_cuda/launch_util.h>
#include <smat/vm/util/specialization_table.h>
#include <smat/vm/util/specialization_typelists.h>
#include <smat/vm/instruction_db.h>

SM_NAMESPACE_BEGIN

template <typename T>
__global__ void kernel_arange(T start, T* dst, usize_t size)
{
	DECL_KERNEL_VARS
	for (usize_t i = (usize_t)bdx*bx+tx; i < size; i += bdx*gdx)
		dst[i] = start + (T)i;
}

template <typename T>
struct execute_arange_typed { // TODO: autotune this
	static void execute(opcode_t opcode, const argument& start, const argument& dst)
	{
		usize_t size = (usize_t)dst.size();
		if (size > 0) {
			launchcfg cfg = make_elemwise_launchcfg(size);
			kernel_arange<<<cfg.gdim,cfg.bdim,cfg.smem,cfg.stream>>>(start.get<T>(),dst.get<T*>(),size);
		}
	}
};

void execute_arange(opcode_t opcode, const argument& start, const argument& dst)
{
	DECL_SPECIALIZATION_TABLE(T_N,execute_fn2,execute_arange_typed);
	specialization_table(dst.dtype)(opcode,start,dst);
}

SM_NAMESPACE_END
