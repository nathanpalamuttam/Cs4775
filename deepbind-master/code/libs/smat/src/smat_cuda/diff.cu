#include "hip/hip_runtime.h"
// Copyright (c) 2015, Andrew Delong and Babak Alipanahi All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
// 
// 1. Redistributions of source code must retain the above copyright notice, this
// list of conditions and the following disclaimer.
// 
// 2. Redistributions in binary form must reproduce the above copyright notice,
// this list of conditions and the following disclaimer in the documentation and/or
// other materials provided with the distribution.
// 
// 3. Neither the name of the copyright holder nor the names of its contributors
// may be used to endorse or promote products derived from this software without
// specific prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
// ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 
// Author's note: 
//     This file was distributed as part of the Nature Biotechnology 
//     supplementary software release for DeepBind. Users of DeepBind
//     are encouraged to instead use the latest source code and binaries 
//     for scoring sequences at
//        http://tools.genes.toronto.edu/deepbind/
// 
#include <smat_cuda/cuda_errors.h>
#include <smat_cuda/cuda_context.h>
#include <smat_cuda/launch_util.h>
#include <smat/vm/util/specialization_table.h>
#include <smat/vm/util/specialization_typelists.h>
#include <smat/vm/instruction_db.h>

SM_NAMESPACE_BEGIN

template <typename T>
__global__ void kernel_diff_y(const T* src, T* dst, usize_t m, usize_t size)
{
	DECL_KERNEL_VARS
	for (usize_t i = (usize_t)bdx*bx+tx; i < size; i += bdx*gdx)
		dst[i] = src[i+m]-src[i];  // could be implemented by oc_sub operation on two views of arg, but this should be marginally faster.
}


template <typename T>
struct execute_diff_typed {
	static void execute(opcode_t opcode, const argument& src, const argument& dst)
	{
		usize_t size = (usize_t)dst.size();
		if (size == 0)
			return;
		if (opcode == oc_diff_x) {
			SM_UNIMPLEMENTED();
		} else if (opcode == oc_diff_y) {
			launchcfg cfg = make_elemwise_launchcfg(size);
			kernel_diff_y<<<cfg.gdim,cfg.bdim,cfg.smem,cfg.stream>>>(src.get<const T*>(),dst.get<T*>(),dst.shape.x,size);
		} else {
			SM_UNREACHABLE();
		}
	}
};

void execute_diff(opcode_t opcode, const argument& src, const argument& dst)
{
	DECL_SPECIALIZATION_TABLE(T_G,execute_fn2,execute_diff_typed);
	specialization_table(src.dtype)(opcode,src,dst);
}

SM_NAMESPACE_END
