#include "hip/hip_runtime.h"
// Copyright (c) 2015, Andrew Delong and Babak Alipanahi All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
// 
// 1. Redistributions of source code must retain the above copyright notice, this
// list of conditions and the following disclaimer.
// 
// 2. Redistributions in binary form must reproduce the above copyright notice,
// this list of conditions and the following disclaimer in the documentation and/or
// other materials provided with the distribution.
// 
// 3. Neither the name of the copyright holder nor the names of its contributors
// may be used to endorse or promote products derived from this software without
// specific prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
// ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 
// Author's note: 
//     This file was distributed as part of the Nature Biotechnology 
//     supplementary software release for DeepBind. Users of DeepBind
//     are encouraged to instead use the latest source code and binaries 
//     for scoring sequences at
//        http://tools.genes.toronto.edu/deepbind/
// 
#include <smat_cuda/cuda_errors.h>
#include <smat_cuda/cuda_context.h>
#include <smat_cuda/launch_util.h>
#include <smat/vm/instruction_db.h>

SM_NAMESPACE_BEGIN

template <typename T>
__global__ void kernel_apply_mask(T* A, const bool* M, unsigned size)
{
	DECL_KERNEL_VARS
	for (unsigned i = bdx*bx + tx; i < size; i += bdx*gdx)
		if (!M[i])
			A[i] = (T)0;
}


void execute_apply_mask(opcode_t opcode, const argument& A, const argument& M)
{
	unsigned size = (unsigned)A.size();
	launchcfg cfg = make_elemwise_launchcfg(size);
	if      (A.dtype == f32) kernel_apply_mask<<<cfg.gdim,cfg.bdim,cfg.smem,cfg.stream>>>(A.get<float* >(),M.get<bool*>(),(unsigned)size);
	else if (A.dtype == f64) kernel_apply_mask<<<cfg.gdim,cfg.bdim,cfg.smem,cfg.stream>>>(A.get<double*>(),M.get<bool*>(),(unsigned)size);
	else { SM_UNIMPLEMENTED(); }
}


SM_NAMESPACE_END
