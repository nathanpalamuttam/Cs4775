#include "hip/hip_runtime.h"
// Copyright (c) 2015, Andrew Delong and Babak Alipanahi All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
// 
// 1. Redistributions of source code must retain the above copyright notice, this
// list of conditions and the following disclaimer.
// 
// 2. Redistributions in binary form must reproduce the above copyright notice,
// this list of conditions and the following disclaimer in the documentation and/or
// other materials provided with the distribution.
// 
// 3. Neither the name of the copyright holder nor the names of its contributors
// may be used to endorse or promote products derived from this software without
// specific prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
// ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 
// Author's note: 
//     This file was distributed as part of the Nature Biotechnology 
//     supplementary software release for DeepBind. Users of DeepBind
//     are encouraged to instead use the latest source code and binaries 
//     for scoring sequences at
//        http://tools.genes.toronto.edu/deepbind/
// 
#include <smat_cuda/cuda_errors.h>
#include <smat_cuda/cuda_context.h>
#include <smat_cuda/launch_util.h>
#include <smat/vm/instruction_db.h>
using namespace sm;

__global__ void kernel_dropoutord_fp_tr(hiprandState_t* state, const uint8_t* X, uint8_t* Z, bool* M, usize_t n, float rate)
{
	DECL_KERNEL_VARS
	unsigned tid = bdx*bx + tx;
	hiprandState local_state = state[tid];
	for (usize_t i = (usize_t)tid; i < n; i += bdx*gdx) {
		bool mask = (X[i] == 255) || (hiprand_uniform(&local_state) >= rate);
		M[i] = mask;
		Z[i] = mask ? X[i] : 254;
	}
	state[tid] = local_state;
}


void launch_dropoutord_fp_tr(hipStream_t stream,
                             const uint8_t* X, uint8_t* Z, bool* M, usize_t n, float rate)
{
	launchcfg cfg = make_elemwise_launchcfg(n);
	kernel_dropoutord_fp_tr<<<cfg.gdim,cfg.bdim,cfg.smem,cfg.stream>>>(thread_cudactx().curand_state(),X,Z,M,n,rate);
}
